#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

template<class Int>
__host__ __device__
static void
inv_lift_orig(Int* p, unsigned int s)
{
	Int x, y, z, w;
	x = *p; p += s;
	y = *p; p += s;
	z = *p; p += s;
	w = *p; p += s;

	y += w >> 1; w -= y >> 1;
	y += w; w <<= 1; w -= y;
	z += x; x <<= 1; x -= z;
	y += z; z <<= 1; z -= y;
	w += x; x <<= 1; x -= w;

	p -= s; *p = w;
	p -= s; *p = z;
	p -= s; *p = y;
	p -= s; *p = x;
}
template<class Int>
__host__ __device__
static void
inv_lift_fix2(Int* p, unsigned int s)
{
	volatile Int x, y, z, w;
	x = *p; p += s;
	y = *p; p += s;
	z = *p; p += s;
	w = *p; p += s;

	y += w >> 1; w -= y >> 1;
	y += w; w <<= 1; w -= y;
	z += x; x <<= 1; x -= z;
	y += z; z <<= 1; z -= y;
	w += x; x <<= 1; x -= w;

	p -= s; *p = w;
	p -= s; *p = z;
	p -= s; *p = y;
	p -= s; *p = x;
}
template<class Int>
__host__ __device__
static void
inv_lift_fix1(Int* p, unsigned int s)
{
	Int x, y, z, w;
	x = *p;
	y = p[s * 1];
	z = p[s * 2];
	w = p[s * 3];

	y += w >> 1; w -= y >> 1;
	y += w; w <<= 1; w -= y;
	z += x; x <<= 1; x -= z;
	y += z; z <<= 1; z -= y;
	w += x; x <<= 1; x -= w;

	p[s * 3] -= s;
	p[s * 2] -= s;
	p[s] -= s;
	p[0] -= s;
}



template<class Int>
__global__
void gpuTest
(
Int *iblock
)
{
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int z = threadIdx.z + blockDim.z*blockIdx.z;
	int idx = z*gridDim.x*blockDim.x*gridDim.y*blockDim.y + y*gridDim.x*blockDim.x + x;
	inv_lift_orig(iblock + idx * 64, 16);


}

template<class Int>
void cpuTest(std::vector<Int> &h_c)
{
	int i = 0;
	//for (i = 0; i < h_c.size(); i++){
	inv_lift_orig(thrust::raw_pointer_cast(h_c.data()) + i * 64, 16);
	//}

}
typedef long long Int;

int main()
{
	const int nx = 256;
	const int ny = 256;
	const int nz = 256;
    const int arraySize = nx*ny*nz;
	const int thread_cnt = arraySize / 64;
	thrust::host_vector<Int> h_cout, h_a;
	h_cout.resize(arraySize);
	h_a.resize(arraySize);

	for (int i = 0; i < arraySize; i++){
		h_a[i] = i;
	}

	thrust::device_vector<Int> d_a, d_c;
	d_a.resize(arraySize);
	d_c.resize(arraySize);

	d_a = h_a;
	d_c = h_a;
    

	gpuTest<Int> << <1,1 >> >(thrust::raw_pointer_cast(d_c.data()));
	//dim3 emax_size(nx / 4, ny / 4, nz / 4);
	//dim3 block_size(8, 8, 8);
	//dim3 grid_size = emax_size;
	//grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;
	//cudaInvXForm<Int> << <block_size, grid_size >> >(thrust::raw_pointer_cast(d_c.data()));
	//hipStreamSynchronize(0);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bitshiftKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
	}

	h_cout = d_c;

	std::vector<Int> h_c;
	h_c.resize(arraySize);
	thrust::copy(h_a.begin(), h_a.end(), h_c.begin());
	cpuTest<Int>(h_c);

	for (int i = 0; i < h_c.size(); i++){
		if (h_c[i] != h_cout[i]){
			std::cout << i << " " << h_c[i] << " " << h_cout[i] << std::endl;
			exit(1);
		}
	}

    return 0;
}

